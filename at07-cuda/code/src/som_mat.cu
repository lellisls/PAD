#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void MatrixAdd_d( float *A, float *B, float *C, int N ) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i * N + j;
  if( ( i < N ) && ( j < N ) ) {
    C[ index ] = A[ index ] + B[ index ];
  }
}

int main( ) {
  float *a_h, *b_h, *c_h; /* pointers to host memory; a.k.a. CPU */
  float *a_d, *b_d, *c_d; /* pointers to device memory; a.k.a. GPU */
  int blocksize = 16, n = 2, i, j, index;
  /* allocate arrays on host */
  a_h = ( float* ) malloc( sizeof( float ) * n * n );
  b_h = ( float* ) malloc( sizeof( float ) * n * n );
  c_h = ( float* ) malloc( sizeof( float ) * n * n );
  /* allocate arrays on device */
  hipMalloc( ( void** ) &a_d, n * n * sizeof( float ) );
  hipMalloc( ( void** ) &b_d, n * n * sizeof( float ) );
  hipMalloc( ( void** ) &c_d, n * n * sizeof( float ) );
  dim3 dimBlock( blocksize, blocksize );
  dim3 dimGrid( ceil( float( n ) / float( dimBlock.x ) ), ceil( float( n ) / float( dimBlock.y ) ) );
  /* initialize the arrays */
  for( j = 0; j < n; j++ ) {
    for( i = 0; i < n; i++ ) {
      index = i * n + j;
      a_h[ index ] = rand( ) % 35;
      b_h[ index ] = rand( ) % 35;
    }
  }
  /* copy and run the code on the device */
  hipMemcpy( a_d, a_h, n * n * sizeof( float ), hipMemcpyHostToDevice );
  hipMemcpy( b_d, b_h, n * n * sizeof( float ), hipMemcpyHostToDevice );
  MatrixAdd_d << < dimGrid, dimBlock >> > ( a_d, b_d, c_d, n );
  hipDeviceSynchronize( );
  hipMemcpy( c_h, c_d, n * n * sizeof( float ), hipMemcpyDeviceToHost );
  /* print out the answer */
  for( j = 0; j < n; j++ ) {
    for( i = 0; i < n; i++ ) {
      index = i * n + j;
      /* This time the array is only 2x2 so we can print it out. */
      printf( "A + B = C: %d %d %f + %f = %f\n", i, j, a_h[ index ], b_h[ index ], c_h[ index ] );
    }
  }
  /* cleanup... */
  free( a_h );
  free( b_h );
  free( c_h );
  hipFree( a_d );
  hipFree( b_d );
  hipFree( c_d );
  return( 0 );
}
